#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
// CUDA headers
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cupti.h>
#include <stdio.h>
#include <iomanip>
#include "gmp/profile.h"

// #define CUPTI_CALL(call)                                                         \
//     do                                                                           \
//     {                                                                            \
//         CUptiResult _status = call;                                              \
//         if (_status != CUPTI_SUCCESS)                                            \
//         {                                                                        \
//             const char *errstr;                                                  \
//             cuptiGetResultString(_status, &errstr);                              \
//             fprintf(stderr, "%s:%d: error: function %s failed with error %s.\n", \
//                     __FILE__, __LINE__, #call, errstr);                          \
//             exit(-1);                                                            \
//         }                                                                        \
//     } while (0)

// Simple CUDA kernel
__global__ void hello_kernel()
{
    printf("Hello World from GPU thread %d!\n", threadIdx.x);
}

// // CUPTI buffer request callback
// void CUPTIAPI bufferRequested(uint8_t **buffer, size_t *size, size_t *maxNumRecords)
// {
//     *size = 16 * 1024;
//     *buffer = (uint8_t *)malloc(*size);
//     *maxNumRecords = 0; // unlimited
// }

// // CUPTI buffer complete callback
// void CUPTIAPI bufferCompleted(hipCtx_t ctx, uint32_t streamId,
//                               uint8_t *buffer, size_t size, size_t validSize)
// {
//     CUptiResult status;
//     CUpti_Activity *record = NULL;

//     // Iterate over records in the buffer
//     while (1)
//     {
//         status = cuptiActivityGetNextRecord(buffer, validSize, &record);
//         if (status == CUPTI_SUCCESS)
//         {
//             if (record->kind == CUPTI_ACTIVITY_KIND_CONCURRENT_KERNEL)
//             {
//                 CUpti_ActivityKernel8 *kernel = (CUpti_ActivityKernel8 *)record;
//                 printf("CUPTI: Kernel \"%s\" launched on stream %u, grid (%u,%u,%u), block (%u,%u,%u)\n",
//                        kernel->name, kernel->streamId,
//                        kernel->gridX, kernel->gridY, kernel->gridZ,
//                        kernel->blockX, kernel->blockY, kernel->blockZ);
//             }
//         }
//         else if (status == CUPTI_ERROR_MAX_LIMIT_REACHED)
//         {
//             printf("CUPTI: Maximum buffer limit reached\n");
//             break;
//         }
//         else
//         {
//             CUPTI_CALL(status);
//         }
//     }

//     // Report dropped records
//     size_t dropped;
//     cuptiActivityGetNumDroppedRecords(ctx, streamId, &dropped);
//     if (dropped != 0)
//     {
//         printf("CUPTI: Dropped %zu activity records\n", dropped);
//     }

//     free(buffer);
// }

__global__ void vecAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
        C[i] = A[i] + B[i];
}

__global__ void multiply(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
        C[i] = A[i] * B[i];
}

__global__ void multiply_complex(float *A, float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        C[i] = A[i] * B[i];
        A[i] = B[i] + C[i];
        B[i] = A[i] * C[i];
    }
}

__global__ void square(float *A, int N)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
    {
        A[i] = A[i] * A[i];
    }
}

__global__ void sumReduction(float *input, float *output, int N)
{
    __shared__ float sdata[256]; // shared memory for partial sums
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    sdata[tid] = (i < N) ? input[i] : 0.0f;
    __syncthreads();

    // reduce within block
    for (int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (tid < s)
        {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0)
        output[blockIdx.x] = sdata[0];
}

#define N 4096 // vector length

void launch_add()
{
    size_t size = N * sizeof(float);

    // Host vectors
    float h_A[N], h_B[N], h_C[N];
    for (int i = 0; i < N; i++)
    {
        h_A[i] = i;
        h_B[i] = i * 10;
    }

    // Device vectors
    float *d_A_1, *d_B_1, *d_C_1;
    float *d_A_2, *d_B_2, *d_C_2;
    float *d_A_3, *d_B_3, *d_C_3;
    float *d_A_4, *d_B_4, *d_C_4;

    hipMalloc((void **)&d_A_1, size);
    hipMalloc((void **)&d_B_1, size);
    hipMalloc((void **)&d_C_1, size);

    hipMalloc((void **)&d_A_2, size);
    hipMalloc((void **)&d_B_2, size);
    hipMalloc((void **)&d_C_2, size);

    hipMalloc((void **)&d_A_3, size);
    hipMalloc((void **)&d_B_3, size);
    hipMalloc((void **)&d_C_3, size);

    hipMalloc((void **)&d_A_4, size);
    hipMalloc((void **)&d_B_4, size);
    hipMalloc((void **)&d_C_4, size);

    // Copy from host to device
    // hipMemcpy(d_A_1, h_A, size, hipMemcpyHostToDevice);
    // hipMemcpy(d_B_1, h_B, size, hipMemcpyHostToDevice);
    // hipMemcpy(d_A_2, h_A, size, hipMemcpyHostToDevice);
    // hipMemcpy(d_B_2, h_B, size, hipMemcpyHostToDevice);
    // hipMemcpy(d_A_3, h_A, size, hipMemcpyHostToDevice);
    // hipMemcpy(d_B_3, h_B, size, hipMemcpyHostToDevice);
    // hipMemcpy(d_A_4, h_A, size, hipMemcpyHostToDevice);
    // hipMemcpy(d_B_4, h_B, size, hipMemcpyHostToDevice);
    // hipMemcpy(d_C_4, h_C, size, hipMemcpyHostToDevice);
    // Launch kernel
    int threadsPerBlock = 512;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    // GmpProfiler::getInstance()->pushRange("launch_add", GmpProfileType::CONCURRENT_KERNEL);
    // vecAdd<<<blocksPerGrid / 2, threadsPerBlock>>>(d_A_1, d_B_1, d_C_1, N / 2);
    // GmpProfiler::getInstance()->popRange("launch_add", GmpProfileType::CONCURRENT_KERNEL);
    // GmpProfiler::getInstance()->pushRange("launch_multiplication", GmpProfileType::CONCURRENT_KERNEL);
    // vecAdd<<<blocksPerGrid / 4, threadsPerBlock>>>(d_A_2, d_B_2, d_C_2, N / 4);
    // GmpProfiler::getInstance()->popRange("launch_multiplication", GmpProfileType::CONCURRENT_KERNEL);
    // GmpProfiler::getInstance()->pushRange("launch_square", GmpProfileType::CONCURRENT_KERNEL);
    // multiply_complex<<<blocksPerGrid, threadsPerBlock>>>(d_A_3, d_B_3, d_C_3, N);
    // GmpProfiler::getInstance()->popRange("launch_square", GmpProfileType::CONCURRENT_KERNEL);
    // GmpProfiler::getInstance()->pushRange("launch_sumReduction", GmpProfileType::CONCURRENT_KERNEL);
    // vecAdd<<<blocksPerGrid / 8, threadsPerBlock>>>(d_A_4, d_B_4, d_C_4, N / 8);
    // GmpProfiler::getInstance()->popRange("launch_sumReduction", GmpProfileType::CONCURRENT_KERNEL);

    // GmpProfiler::getInstance()->pushRange("launch_add");
    // vecAdd<<<blocksPerGrid / 2, threadsPerBlock>>>(d_A_1, d_B_1, d_C_1, N / 2);
    // vecAdd<<<blocksPerGrid / 4, threadsPerBlock>>>(d_A_2, d_B_2, d_C_2, N / 4);
    // multiply_complex<<<blocksPerGrid, threadsPerBlock>>>(d_A_3, d_B_3, d_C_3, N);
    // GmpProfiler::getInstance()->popRange();
    hipDeviceSynchronize();

    hello_kernel<<<1, 4>>>();
    GmpProfiler::getInstance()->pushRange("allallall");
    GmpProfiler::getInstance()->pushRange("all", GmpProfileType::CONCURRENT_KERNEL);
    vecAdd<<<blocksPerGrid / 2, threadsPerBlock>>>(d_A_1, d_B_1, d_C_1, N / 2);
    vecAdd<<<blocksPerGrid / 4, threadsPerBlock>>>(d_A_2, d_B_2, d_C_2, N / 4);
    multiply_complex<<<blocksPerGrid, threadsPerBlock>>>(d_A_3, d_B_3, d_C_3, N);
    vecAdd<<<blocksPerGrid / 8, threadsPerBlock>>>(d_A_1, d_B_1, d_C_1, N / 8);
    GmpProfiler::getInstance()->popRange("all", GmpProfileType::CONCURRENT_KERNEL);
    GmpProfiler::getInstance()->popRange();
    hipDeviceSynchronize();
    // Copy result back

    // Cleanup
    hipFree(d_A_1);
    hipFree(d_B_1);
    hipFree(d_C_1);
    hipFree(d_A_2);
    hipFree(d_B_2);
    hipFree(d_C_2);
    hipFree(d_A_3);
    hipFree(d_B_3);
    hipFree(d_C_3);
}

int main()
{
    // CUPTI_CALL(cuptiActivityEnable(CUPTI_ACTIVITY_KIND_CONCURRENT_KERNEL));

    // CUPTI_CALL(cuptiActivityRegisterCallbacks(bufferRequested, bufferCompleted));

    hello_kernel<<<1, 4>>>();
    int curr_pass = 0;
    GmpProfiler::getInstance()->init();
    printf("Starting profiling runs...\n");
#ifdef USE_CUPTI
// while (GmpProfiler::getInstance()->isAllPassSubmitted() == false)
// {
#endif
    printf("current pass: %zu\n", curr_pass++);
    GmpProfiler::getInstance()->startRangeProfiling();
    for (int i = 0; i < 1000; i++)
    {
        launch_add();
    }
    GmpProfiler::getInstance()->stopRangeProfiling();
#ifdef USE_CUPTI
// }
#endif
    hipDeviceSynchronize();
    GmpProfiler::getInstance()->decodeCounterData();
    GmpProfiler::getInstance()->printProfilerRanges();

    CUPTI_CALL(cuptiActivityFlushAll(1));

    CUPTI_CALL(cuptiActivityDisable(CUPTI_ACTIVITY_KIND_CONCURRENT_KERNEL));

    return 0;
}
